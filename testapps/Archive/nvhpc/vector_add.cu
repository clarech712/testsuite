#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vector_add(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 100;  // Size of vectors
    float *a, *b, *c;  // Host pointers
    float *d_a, *d_b, *d_c;  // Device pointers

    size_t bytes = n * sizeof(float);

    // Allocate memory on host
    a = (float*)malloc(bytes);
    b = (float*)malloc(bytes);
    c = (float*)malloc(bytes);

    // Initialize vectors on host
    for (int i = 0; i < n; i++) {
        a[i] = float(i);
        b[i] = float(i) * 2.0f;
    }

    // Allocate memory on device
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy vectors from host to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    // Define block size and grid size
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    // Launch the kernel
    vector_add<<<grid_size, block_size>>>(d_a, d_b, d_c, n);

    // Copy result from device to host
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result: ";
    for (int i = 0; i < n; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
